
#include <hip/hip_runtime.h>
struct cuTimer
{
    hipEvent_t _start;
    hipEvent_t _end;
    float elapsedTime;

    void start()
    {
        hipEventCreate(&_start);
        hipEventCreate(&_end);
        hipEventRecord(_start);
    }

    void stop()
    {
        hipEventRecord(_end);
        hipEventSynchronize(_end);
        hipEventElapsedTime(&elapsedTime, _start, _end);
    }

    ~cuTimer()
    {
        hipEventDestroy(_start);
        hipEventDestroy(_end);
    }
};